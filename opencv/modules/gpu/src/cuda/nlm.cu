#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Copyright (C) 1993-2011, NVIDIA Corporation, all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"

#include "opencv2/gpu/device/vec_traits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"

using namespace cv::gpu;

typedef unsigned char uchar;
typedef unsigned short ushort;

//////////////////////////////////////////////////////////////////////////////////
//// Non Local Means Denosing

namespace cv { namespace gpu { namespace device
{
    namespace imgproc
    {
        __device__ __forceinline__ float norm2(const float& v) { return v*v; }
        __device__ __forceinline__ float norm2(const float2& v) { return v.x*v.x + v.y*v.y; }
        __device__ __forceinline__ float norm2(const float3& v) { return v.x*v.x + v.y*v.y + v.z*v.z; }
        __device__ __forceinline__ float norm2(const float4& v) { return v.x*v.x + v.y*v.y + v.z*v.z  + v.w*v.w; }

        template<typename T, typename B>
        __global__ void nlm_kernel(const PtrStepSz<T> src, PtrStep<T> dst, const B b, int search_radius, int block_radius, float h2_inv_half)
        {
            typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type value_type;

            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x >= src.cols || y >= src.rows)
                return;

            float block_radius2_inv = -1.f/(block_radius * block_radius);

            value_type sum1 = VecTraits<value_type>::all(0);
            float sum2 = 0.f;

            if (x - search_radius - block_radius >=0        && y - search_radius - block_radius >=0 &&
                x + search_radius + block_radius < src.cols && y + search_radius + block_radius < src.rows)
            {

                for(float cy = -search_radius; cy <= search_radius; ++cy)
                    for(float cx = -search_radius; cx <= search_radius; ++cx)
                    {
                        float color2 = 0;
                        for(float by = -block_radius; by <= block_radius; ++by)
                            for(float bx = -block_radius; bx <= block_radius; ++bx)
                            {
                                value_type v1 = saturate_cast<value_type>(src(y +      by, x +      bx));
                                value_type v2 = saturate_cast<value_type>(src(y + cy + by, x + cx + bx));
                                color2 += norm2(v1 - v2);
                            }

                        float dist2 = cx * cx + cy * cy;
                        float w = __expf(color2 * h2_inv_half + dist2 * block_radius2_inv);

                        sum1 = sum1 + saturate_cast<value_type>(src(y + cy, x + cy)) * w;
                        sum2 += w;
                    }
            }
            else
            {
                for(float cy = -search_radius; cy <= search_radius; ++cy)
                    for(float cx = -search_radius; cx <= search_radius; ++cx)
                    {
                        float color2 = 0;
                        for(float by = -block_radius; by <= block_radius; ++by)
                            for(float bx = -block_radius; bx <= block_radius; ++bx)
                            {
                                value_type v1 = saturate_cast<value_type>(b.at(y +      by, x +      bx, src.data, src.step));
                                value_type v2 = saturate_cast<value_type>(b.at(y + cy + by, x + cx + bx, src.data, src.step));
                                color2 += norm2(v1 - v2);
                            }

                        float dist2 = cx * cx + cy * cy;
                        float w = __expf(color2 * h2_inv_half + dist2 * block_radius2_inv);

                        sum1 = sum1 + saturate_cast<value_type>(b.at(y + cy, x + cy, src.data, src.step)) * w;
                        sum2 += w;
                    }

            }

            dst(y, x) = saturate_cast<T>(sum1 / sum2);

        }

        template<typename T, template <typename> class B>
        void nlm_caller(const PtrStepSzb src, PtrStepSzb dst, int search_radius, int block_radius, float h, hipStream_t stream)
        {
            dim3 block (32, 8);
            dim3 grid (divUp (src.cols, block.x), divUp (src.rows, block.y));

            B<T> b(src.rows, src.cols);

            float h2_inv_half = -0.5f/(h * h * VecTraits<T>::cn);

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(nlm_kernel<T), B<T> >, hipFuncCachePreferL1) );
            nlm_kernel<<<grid, block>>>((PtrStepSz<T>)src, (PtrStepSz<T>)dst, b, search_radius, block_radius, h2_inv_half);
            cudaSafeCall ( hipGetLastError () );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template<typename T>
        void nlm_bruteforce_gpu(const PtrStepSzb& src, PtrStepSzb dst, int search_radius, int block_radius, float h, int borderMode, hipStream_t stream)
        {
            typedef void (*func_t)(const PtrStepSzb src, PtrStepSzb dst, int search_radius, int block_radius, float h, hipStream_t stream);

            static func_t funcs[] = 
            {
                nlm_caller<T, BrdReflect101>,
                nlm_caller<T, BrdReplicate>,
                nlm_caller<T, BrdConstant>,
                nlm_caller<T, BrdReflect>,
                nlm_caller<T, BrdWrap>,
            };
            funcs[borderMode](src, dst, search_radius, block_radius, h, stream);
        }

        template void nlm_bruteforce_gpu<uchar>(const PtrStepSzb&, PtrStepSzb, int, int, float, int, hipStream_t);
        template void nlm_bruteforce_gpu<uchar3>(const PtrStepSzb&, PtrStepSzb, int, int, float, int, hipStream_t);
    }
}}}
